#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

#define CEIL_DIV(x, y) ((x + y - 1) / y)

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
  }
}

__device__ int2 divide_work(int n_jobs, int n_workers, int worker_idx) {
  // Each worker will do a continuous slice of either n_jobs / n_workers
  // or ceil_div(n_jobs, n_workers). The return value is an int2 representing
  // a half open interval of jobs for the worker to perform (perform jobs
  // i for a <= i < b)

  int cd = CEIL_DIV(n_jobs, n_workers);
  int d = n_jobs / n_workers;

  int doing_cd = n_jobs % n_workers;

  int2 retval;
  if (worker_idx < doing_cd) {
    retval.x = worker_idx * cd;
    retval.y = retval.x + cd;
  } else {
    retval.x = doing_cd * cd + (worker_idx - doing_cd) * d;
    retval.y = retval.x + d;
  }

  return retval;
}

__device__ int2 compute_warp_start_stop(int block_idx, int warp_idx,
					int n_blocks, int n_steps) {
  int2 block_ss = divide_work(n_steps, n_blocks, block_idx);
  int block_start = block_ss.x;
  int block_stop = block_ss.y;
  int block_jobs = block_stop - block_start;

  int2 warp_ss = divide_work(block_jobs, 32, warp_idx);
  int warp_start = block_start + warp_ss.x;
  int warp_stop = block_start + warp_ss.y;

  int2 retval;
  retval.x = warp_start;
  retval.y = warp_stop;
  return retval;
}

// decay storage, h_storage:
//   each a n_dims x 33 x n_blocks matrix on GPU with 33rd column for block reduction
__global__ void reduction_kernel(float *decays, float *impulses,
				 float *initial_state,
				 float *_decay_storage, float *_h_storage,
				 int n_dims, int n_steps) {
  int warp = threadIdx.x / 32;
  int lane = threadIdx.x % 32;

  float *decay_storage = &_decay_storage[blockIdx.x * 33 * n_dims];
  float *h_storage = &_h_storage[blockIdx.x * 33 * n_dims];

  int2 start_stop = compute_warp_start_stop(blockIdx.x, warp, gridDim.x, n_steps);
  int warp_start = start_stop.x;
  int warp_stop = start_stop.y;

  /*
  * Reduce within warps.
  * After this loop exits, the storage arrays should contain the reduction
  * from warp_start to warp_stop (including initial state) at index
  * (feature_idx, warp, block).
  */
  for (int i = lane; i < n_dims; i += 32) {
    float cum_decay = 1.0;
    float h = 0.0;
    if (blockIdx.x == 0 && warp == 0 && initial_state != NULL) {
      h = initial_state[i];
    }

    for (int t = warp_start; t < warp_stop; t++) {
      cum_decay *= decays[i + t * n_dims];
      h = decays[i + t * n_dims] * h + impulses[i + t * n_dims];
    }

    // TODO: store into shared memory, work in shared memory sized blocks
    // store into global memory
    decay_storage[i + warp * n_dims] = cum_decay;
    h_storage[i + warp * n_dims] = h;
  }

  __syncthreads();

  /*
   * Reduce over warps.
   * After this loop exits, the storage arrays should contain the reduction
   * from block_start to block_finish (including initial state) at index
   * (feature_idx, 32, block).
   */
  // TODO: parallel reduction (or scan). Need to worry about changing the warp
  //       reduction values (as I use them again later)
  for (int i = lane + 32 * warp; i < n_dims; i += blockDim.x) {
    float cum_decay = 1.0;
    float h = 0.0;
    for (int t = 0; t < 32; t++) {
      cum_decay *= decay_storage[i + t * n_dims];
      h = decay_storage[i + t * n_dims] * h + h_storage[i + t * n_dims];
    }
    decay_storage[i + 32 * n_dims] = cum_decay;
    h_storage[i + 32 * n_dims] = h;
  }
}

__global__ void block_scan_kernel(float *decay_storage, float *h_storage,
				  int n_dims, int n_blocks) {
  /*
   * Scan over blocks.
   * After this loop exits, the storage arrays should contain the cumulative sum
   * from block_idx 0 to i (inclusive) at index (feature_idx, 32, i)
   * This means (feature_idx, 32, 2) contains the reduction of blocks 0, 1, and 2.
   */
  // TODO: parallel scan (tricky because number of blocks isn't necessarily
  //       smaller than number of warps that can fit in a single block)
  for (int i = threadIdx.x + blockIdx.x * blockDim.x;
       i < n_dims;
       i += blockDim.x * gridDim.x) {

    for (int t = 1; t < n_blocks; t++) {
      int cur_idx = i + 32 * n_dims + t * 33 * n_dims;
      int prev_idx = i + 32 * n_dims + (t - 1) * 33 * n_dims;

      // TODO: remove unneccessary reads from global memory (prev_idx accesses)
      h_storage[cur_idx] = decay_storage[cur_idx] * h_storage[prev_idx] + h_storage[cur_idx];
      decay_storage[cur_idx] *= decay_storage[prev_idx];
    }
  }
}

__global__ void warp_scan_kernel(float *decays, float *impulses,
				 float *initial_state, float *out,
				 float *decay_storage, float *h_storage,
				 int n_dims, int n_steps) {
  int warp = threadIdx.x / 32;
  int lane = threadIdx.x % 32;

  // Note: Due to the index ordering of the storage arrays, the following
  // indices are equivalent:
  //
  // i + (t - 1) * n_dims + blockIdx.x * 33 * n_dims
  // i + 32 * n_dims + (blockIdx.x - 1) * 33 * n_dims
  //
  // when t is 0. This means something that looks like negative indexing
  // (t-1) can be used to safely access the stored value for the previous
  // warp (even if the previous warp belonged to the previous block).

  /*
   * Scan over warps.
   * After this loop executes, the storage arrays should contain the cumulative
   * sum from the beginning of sequence (including initial condition) up to
   * and including the indexed warp and block.
   */
  // TODO: parallel scan
  for (int i = lane + 32 * warp; i < n_dims; i += blockDim.x) {
    for (int t = 0; t < 32; t++) {
      if (t == 0 && blockIdx.x == 0) {
        // the reduction over warp 0 (including initial condition) is correct val
        // for scan, so there's no work to do
        continue;
      }

      int cur_idx = i + t * n_dims + blockIdx.x * 33 * n_dims;
      int prev_idx = i + (t - 1) * n_dims + blockIdx.x * 33 * n_dims;
      h_storage[cur_idx] = decay_storage[cur_idx] * h_storage[prev_idx] + h_storage[cur_idx];
      decay_storage[cur_idx] *= decay_storage[prev_idx];
    }
  }

  __syncthreads();

  int2 start_stop = compute_warp_start_stop(blockIdx.x, warp, gridDim.x, n_steps);
  int warp_start = start_stop.x;
  int warp_stop = start_stop.y;

  /*
   * Scan within warps.
   * This loop writes to the output array. Each warp reads in it's initial state
   * (either from the "initial_state" or the storage arrays) and then writes
   * to output for indices warp_start up to warp_stop.
   */
  for (int i = lane; i < n_dims; i += 32) {
    float h = 0.0;
    if (blockIdx.x == 0 && warp == 0) {
      if (initial_state != NULL) {
	h = initial_state[i];
      }
    } else {
      h = h_storage[i + (warp - 1) * n_dims + blockIdx.x * 33 * n_dims];
    }

    for (int t = warp_start; t < warp_stop; t++) {
      h = decays[i + t * n_dims] * h + impulses[i + t * n_dims];
      out[i + t * n_dims] = h;
    }
  }
}

__global__ void serial_linear_recurrence(float *decays, float *impulses,
                                         float *initial_state, float *out,
                                         int n_dims, int n_steps) {
  // computes h_t = lambda_t h{t-1} + x_t

  for (int dim_idx = threadIdx.x + blockIdx.x * blockDim.x;
       dim_idx < n_dims;
       dim_idx += blockDim.x * gridDim.x) {
    float val = initial_state[dim_idx];

    for (int step = 0; step < n_steps; step++) {
      int idx = dim_idx + step * n_dims;
      val = decays[idx] * val + impulses[idx];
      out[idx] = val;
    }
  }
}

extern "C" {
/*
 * This is the main method for the prefix sum kernels.
 * decays, impulses, out:
 *   each a n_dims x n_steps column major matrix located on GPU
 * initial_state:
 *   array of size n_dims located on GPU
 */
void compute_linear_recurrence(float *decays, float *impulses, float *initial_state,
			       float *out, int n_dims, int n_steps) {

  // TODO: query
  int n_SMs = 15;
  int n_blocks_per_sm = 2;

  // we want at least 32 elements per block, but no reason to run
  // with more than the maximum number of concurrent blocks
  int n_blocks = min(CEIL_DIV(n_steps, 32), n_SMs * n_blocks_per_sm);

  // TODO: make user pass in working memory? This allows integration
  //       with CNMeM (used by Theano)
  int reduction_mem_sz = 2 * n_blocks * 33 * n_dims * sizeof(float);
  float *d_reduction_mem;
  gpuErrChk(hipMalloc(&d_reduction_mem, reduction_mem_sz));
  float *d_decay_storage = &d_reduction_mem[0 * n_blocks * 33 * n_dims];
  float *d_h_storage = &d_reduction_mem[1 * n_blocks * 33 * n_dims];

  // TODO: run kernels on non-default stream?
  reduction_kernel<<<n_blocks, 1024>>>(decays, impulses, initial_state,
				       d_decay_storage, d_h_storage,
				       n_dims, n_steps);

  block_scan_kernel<<<n_blocks, 1024>>>(d_decay_storage, d_h_storage,
					n_dims, n_blocks);

  warp_scan_kernel<<<n_blocks, 1024>>>(decays, impulses,
				       initial_state, out,
				       d_decay_storage, d_h_storage,
				       n_dims, n_steps);

  gpuErrChk(hipFree(d_reduction_mem));
}

void compute_serial_linear_recurrence(float *decays, float *impulses,
                                      float *initial_state, float *out,
                                      int n_dims, int n_steps) {
  // TODO: query
  int n_SMs = 15;
  int n_blocks_per_sm = 2;

  int n_blocks = n_SMs * n_blocks_per_sm;
  serial_linear_recurrence<<<n_blocks, 1024>>>(decays, impulses, initial_state,
                                               out, n_dims, n_steps);
}
}

void test() {
  int n_dims = 100;
  int n_steps = 1000000;
  int n_elements = n_dims * n_steps;

  float *decays = (float *) calloc(n_elements, sizeof(float));
  for (int i = 0; i < n_elements; i++) {
    decays[i] = .999;
  }
  float *d_decays;
  gpuErrChk(hipMalloc(&d_decays, n_elements * sizeof(float)));
  gpuErrChk(hipMemcpy(d_decays, decays, n_elements * sizeof(float),
		       hipMemcpyHostToDevice));

  float *impulses = (float *) calloc(n_elements, sizeof(float));
  for (int i = 0; i < n_dims; i++) {
    impulses[i + 0 * n_dims] = 2.0;
  }
  float *d_impulses;
  gpuErrChk(hipMalloc(&d_impulses, n_elements * sizeof(float)));
  gpuErrChk(hipMemcpy(d_impulses, impulses,
		       n_elements * sizeof(float), hipMemcpyHostToDevice));

  float *out = (float *) calloc(n_elements, sizeof(float));
  float *d_out;
  gpuErrChk(hipMalloc(&d_out, n_elements * sizeof(float)));
  gpuErrChk(hipMemset(d_out, 0, n_elements * sizeof(float)));

  compute_linear_recurrence(d_decays, d_impulses, NULL, d_out, n_dims, n_steps);
  gpuErrChk(hipMemcpy(out, d_out, n_elements * sizeof(float),
		       hipMemcpyDeviceToHost));

  gpuErrChk(hipFree(d_decays));
  gpuErrChk(hipFree(d_impulses));
  gpuErrChk(hipFree(d_out));
}
